#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <ctime>
#include "triangle.h"
#include <vector>
#include <stdio.h>
#include <iostream>
#include "delauney.h"

#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/imgproc/imgproc.hpp"

#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
using namespace std;

uint8_t* device_img = NULL;
float* device_grad = NULL;
Point* device_seeds = NULL;
Triangle* device_triangles = NULL;
int* device_owner = NULL;
int* mutex;
int* numTriangles;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int NextPower2_CPU(int v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

int Index_CPU(Point p, int col)
{
    return p.y * col + p.x;
}

__device__ __inline__ bool InBound(Point p, int row, int col)
{
    return (0 <= p.x && p.x < col && 0<=p.y && p.y < row);
}

__device__ __inline__ int Index(Point p, int col)
{
    return p.y * col + p.x;
}

__global__ void try_kernel()
{
    Triangle triangle_1;
    Triangle triangle(Point(0, 0), Point(0, 3), Point(3, 3));
    Point center = triangle.center();
    printf("%d, %d\n", center.x, center.y);
}

__global__ void voronoi_kernel(Point* device_seeds, int* device_owner, int stepsize, int rows, int cols)
{   
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return;

    Point dir[] = {Point(0,1), Point(0, -1), Point(1, 0), Point(-1, 0),
                    Point(1,1), Point(1, -1), Point(-1, 1), Point(-1,-1)};

    Point now_point(c, r);
    for(Point now_dir: dir)
    {
        Point now_looking = now_point + now_dir * stepsize;
        if(!InBound(now_looking, rows, cols))
            continue;

        if(device_owner[Index(now_looking, cols)] == -1)
            continue;

        int cand_dist = dist(device_seeds[device_owner[Index(now_looking, cols)]], now_point);
        int now_owner = device_owner[Index(now_point, cols)];

        if(now_owner == -1 || cand_dist < dist(device_seeds[now_owner], now_point))
            device_owner[Index(now_point, cols)] = device_owner[Index(now_looking, cols)];
    }
}

__device__ void count_colors(Point now_point, int device_owner[], int colors[4], int &numColors, int cols)
{
    Point neighbor_dir[] = {Point(1, 0), Point(0, 1), Point(1, 1)};

    colors[0] = device_owner[Index(now_point, cols)];
    numColors = 1;
    for(Point now_dir: neighbor_dir)
    {
        Point next_point = now_point + now_dir;
        int newColor = device_owner[Index(next_point, cols)];
        bool exist = false;
        for (int i = 0; i < numColors; i++)
        {
            if (newColor == colors[i])
            {
                exist = true;
                break;
            }
        }
        if (!exist)
        {
            colors[numColors] = newColor;
            numColors ++;
        }
    }
}

// Parallelize by pixel, tell the caller the #triangle at (c, r)
__global__ void count_triangle_kernel(int* device_owner, int rows, int cols, int* triangle_count)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols-1 || r >= rows-1)
        return;

    Point now_point(c, r);

    int colors[4] = {-1, -1, -1, -1};
    int numColors;
    count_colors(now_point, device_owner, colors, numColors, cols);

    if(numColors == 3)
        triangle_count[Index(now_point, cols)] = 1;
    else if(numColors == 4)
        triangle_count[Index(now_point, cols)] = 2;
    else
        triangle_count[Index(now_point, cols)] = 0;
}

// put triangle to device_triangles
__global__ void triangle_kernel(Point* device_seeds, int* device_owner, Triangle* device_triangles, int rows, int cols, int* device_sum_triangles)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols-1 || r >= rows-1)
        return;

    Point now_point(c, r);

    int colors[4] = {-1, -1, -1, -1};
    int numColors;
    count_colors(now_point, device_owner, colors, numColors, cols);

    int index = Index(now_point, cols);
    int prev_triangle_cnt = (index == 0)? 0: device_sum_triangles[index-1];

    if(numColors == 3)
    {
        Triangle triangle;
        for(int i=0; i<3; i++)
        {
            triangle.points[i] = device_seeds[colors[i]];
        }

        device_triangles[prev_triangle_cnt] = triangle;
    }
    else if(numColors == 4)
    {
        Triangle triangle1(device_seeds[device_owner[Index(now_point, cols)]],
                           device_seeds[device_owner[Index(now_point + Point(1, 0), cols)]],
                           device_seeds[device_owner[Index(now_point + Point(0, 1), cols)]]);

        Triangle triangle2(device_seeds[device_owner[Index(now_point + Point(1, 0), cols)]],
                           device_seeds[device_owner[Index(now_point + Point(0, 1), cols)]],
                           device_seeds[device_owner[Index(now_point + Point(1, 1), cols)]]);

        device_triangles[prev_triangle_cnt] = triangle1;
        device_triangles[prev_triangle_cnt+1] = triangle2;
    }
    return;
}


__global__ void get_grad_kernel(uint8_t* device_img, float* device_grad, int rows, int cols)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return;

    if (r > 0 && c > 0 && r < rows - 1 && c < cols - 1) // inside the image
    {
        float grad_x = -(float)device_img[(r - 1) * cols + c - 1] + (float)device_img[(r - 1) * cols + c + 1]
                       - 2 * (float)device_img[r * cols + c - 1]  + 2 * (float)device_img[r * cols + c + 1]
                       -(float)device_img[(r + 1) * cols + c - 1] + (float)device_img[(r + 1) * cols + c + 1];
        grad_x = abs(grad_x);

        float grad_y = -(float)device_img[(r - 1) * cols + c - 1] - 2 * (float)device_img[(r - 1) * cols + c] - (float)device_img[(r - 1) * cols + c + 1]
                       +(float)device_img[(r + 1) * cols + c - 1] + 2 * (float)device_img[(r + 1) * cols + c] + (float)device_img[(r + 1) * cols + c + 1];
        grad_y = abs(grad_y);

        float grad = grad_x / 2.0 + grad_y / 2.0;
        device_grad[r * cols + c] = grad;
    }

    else // set the boundary values to 0
    {
        device_grad[r * cols + c] = 0;
    }
}


void PrintDevice()
{
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}


cv::Mat getGradGPU(cv::Mat &img)
{
    int rows = img.rows;
    int cols = img.cols;
    int numPixel = rows * cols;

    cv::Mat imgGray;
    imgGray.create(rows, cols, CV_8UC1);
    cv::cvtColor(img, imgGray, CV_BGR2GRAY);

    hipMalloc(&device_img, sizeof(uint8_t)*numPixel);
    hipMalloc(&device_grad, sizeof(float)*numPixel);
    hipMemcpy(device_img, imgGray.data, sizeof(uint8_t)*numPixel, hipMemcpyHostToDevice);

    unsigned int n = 32;
    dim3 blockDim(n, n);
    dim3 gridDim((cols + n - 1) / n, (rows + n - 1) / n);
    get_grad_kernel<<<gridDim, blockDim>>>(device_img, device_grad, rows, cols);
    gpuErrchk(hipDeviceSynchronize());

    cv::Mat grad;
    grad.create(rows, cols, CV_32FC1);
    hipMemcpy(grad.data, device_grad, sizeof(float)*numPixel, hipMemcpyDeviceToHost);

    hipFree(device_grad);
    
    return grad;
}


vector<Triangle> DelauneyGPU(Point* seeds, int numSeeds, int* owner, int rows, int cols)
{
    PrintDevice();

    // put seeds on the graph
    for(int i=0; i<numSeeds; i++)
    {
        Point seed = seeds[i];
        owner[Index_CPU(seed, cols)] = i;
    }

    // define grid and block size
    unsigned int n = 32;
    dim3 blockDim(n, n);
    dim3 gridDim((cols + n - 1) / n, (rows + n - 1) / n);


    // transfer seeds and owner to device
    clock_t mem_start = clock();

    hipMalloc(&device_seeds, sizeof(Point)*numSeeds);
    hipMalloc(&device_owner, sizeof(int)*rows*cols);
    printf("Mem Time: %lf\n", (clock() - mem_start) / (double)(CLOCKS_PER_SEC / 1000));

    hipMemcpy(device_seeds, seeds, sizeof(Point)*numSeeds, hipMemcpyHostToDevice);


    hipMemcpy(device_owner, owner, sizeof(int)*rows*cols, hipMemcpyHostToDevice);

    clock_t comp_start = clock();

    // Step1 : find Voronoi graph
    int start_stepsize = NextPower2_CPU(min(rows, cols)) / 2;
    for(int stepsize = start_stepsize; stepsize>=1; stepsize /= 2)
    {
        voronoi_kernel<<<gridDim, blockDim>>>(device_seeds, device_owner, stepsize, rows, cols);
        gpuErrchk(hipDeviceSynchronize());
    }

    // copy owner data to CPU
    hipMemcpy(owner, device_owner, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);

    // Step 2: find the number of triangles
    int* device_triangle_cnts;
    hipMalloc(&device_triangle_cnts, sizeof(int) * rows * cols);

    count_triangle_kernel<<<gridDim, blockDim>>>(device_owner, rows, cols, device_triangle_cnts);
    gpuErrchk(hipDeviceSynchronize());

    // Step 3: prefix sum #triangles
    int* device_sum_triangles;
    hipMalloc(&device_sum_triangles, sizeof(int) * rows * cols);
    thrust::inclusive_scan(thrust::device, device_triangle_cnts, device_triangle_cnts + rows*cols, device_sum_triangles);

    // Step 4: build the triangles
    Triangle* device_triangles;
    int num_triangles;
    hipMemcpy(&num_triangles, &device_sum_triangles[rows*cols-1], sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&device_triangles, sizeof(Triangle) * num_triangles);
    triangle_kernel<<<gridDim, blockDim>>>(device_seeds, device_owner, device_triangles, rows, cols, device_sum_triangles);
    gpuErrchk(hipDeviceSynchronize());

    // copy triangle data to CPU
    Triangle triangles[num_triangles];
    hipMemcpy(triangles, device_triangles, sizeof(Triangle)*num_triangles, hipMemcpyDeviceToHost);

    vector<Triangle> ret(triangles, triangles + num_triangles);

    // free
    hipFree(device_seeds);
    hipFree(device_owner);
    hipFree(device_triangle_cnts);
    hipFree(device_sum_triangles);
    hipFree(device_triangles);

    cout<<"Core computation time: "<< (clock() - comp_start) / (double)(CLOCKS_PER_SEC / 1000) <<endl;

    return ret;
}
