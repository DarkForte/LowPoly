#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <ctime>
#include "triangle.h"
#include <vector>
#include <stdio.h>
#include <iostream>
#include "delauney.h"
#include "cycleTimer.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/imgproc/imgproc.hpp"

#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
using namespace std;

uint8_t* device_img = NULL;
float* device_grad = NULL;
Point* device_seeds = NULL;
Point* device_ownerMap = NULL;
int* device_owner = NULL;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int NextPower2_CPU(int v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

int Index_CPU(Point p, int col)
{
    return p.y * col + p.x;
}

__device__ __inline__ bool InBound(Point p, int row, int col)
{
    return (0 <= p.x && p.x < col && 0<=p.y && p.y < row);
}

__device__ __inline__ int Index(Point p, int col)
{
    return p.y * col + p.x;
}

__global__ void voronoi_kernel(Point* device_owner, int stepsize, int rows, int cols)
{   
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return;

    Point dir[] = {Point(0,1), Point(0, -1), Point(1, 0), Point(-1, 0),
                    Point(1,1), Point(1, -1), Point(-1, 1), Point(-1,-1)};

    Point now_point(c, r);
    for(Point now_dir: dir)
    {
        Point now_looking = now_point + now_dir * stepsize;
        if(!InBound(now_looking, rows, cols))
            continue;

        if(device_owner[Index(now_looking, cols)].isInvalid())
            continue;

        int cand_dist = dist(device_owner[Index(now_looking, cols)], now_point);
        Point now_owner = device_owner[Index(now_point, cols)];

        if(now_owner.isInvalid() || cand_dist < dist(now_owner, now_point))
            device_owner[Index(now_point, cols)] = device_owner[Index(now_looking, cols)];
    }
}

// Color = a seed
__device__ void count_colors(Point now_point, Point device_owner[], Point colors[4], int &numColors, int cols)
{
    Point neighbor_dir[] = {Point(1, 0), Point(0, 1), Point(1, 1)};

    colors[0] = device_owner[Index(now_point, cols)];
    numColors = 1;
    for(Point now_dir: neighbor_dir)
    {
        Point next_point = now_point + now_dir;
        Point newColor = device_owner[Index(next_point, cols)];
        bool exist = false;
        for (int i = 0; i < numColors; i++)
        {
            if (newColor == colors[i])
            {
                exist = true;
                break;
            }
        }
        if (!exist)
        {
            colors[numColors] = newColor;
            numColors ++;
        }
    }
}

// Parallelize by pixel, tell the caller the #triangle at (c, r)
__global__ void count_triangle_kernel(Point* device_owner, int rows, int cols, int* triangle_count)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols-1 || r >= rows-1)
        return;

    Point now_point(c, r);

    Point colors[4] = {Point(-1, -1), Point(-1, -1), Point(-1, -1), Point(-1, -1)};
    int numColors;
    count_colors(now_point, device_owner, colors, numColors, cols);

    if(numColors == 3)
        triangle_count[Index(now_point, cols)] = 1;
    else if(numColors == 4)
        triangle_count[Index(now_point, cols)] = 2;
    else
        triangle_count[Index(now_point, cols)] = 0;
}

// put triangle to device_triangles
__global__ void triangle_kernel(Point* device_owner, Triangle* device_triangles, int rows, int cols, int* device_sum_triangles)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols-1 || r >= rows-1)
        return;

    Point now_point(c, r);

    Point colors[4] = {Point(-1, -1), Point(-1, -1), Point(-1, -1), Point(-1, -1)};
    int numColors;
    count_colors(now_point, device_owner, colors, numColors, cols);

    int index = Index(now_point, cols);
    int prev_triangle_cnt = (index == 0)? 0: device_sum_triangles[index-1];

    if(numColors == 3)
    {
        Triangle triangle;
        for(int i=0; i<3; i++)
        {
            triangle.points[i] = colors[i];
        }

        device_triangles[prev_triangle_cnt] = triangle;
    }
    else if(numColors == 4)
    {
        Triangle triangle1(device_owner[Index(now_point, cols)],
                           device_owner[Index(now_point + Point(1, 0), cols)],
                           device_owner[Index(now_point + Point(0, 1), cols)]);

        Triangle triangle2(device_owner[Index(now_point + Point(1, 0), cols)],
                           device_owner[Index(now_point + Point(0, 1), cols)],
                           device_owner[Index(now_point + Point(1, 1), cols)]);

        device_triangles[prev_triangle_cnt] = triangle1;
        device_triangles[prev_triangle_cnt+1] = triangle2;
    }
    return;
}


__global__ void get_grad_kernel(uint8_t* device_img, float* device_grad, int rows, int cols)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return;

    if (r > 0 && c > 0 && r < rows - 1 && c < cols - 1) // inside the image
    {
        float grad_x = -(float)device_img[(r - 1) * cols + c - 1] + (float)device_img[(r - 1) * cols + c + 1]
                       - 2 * (float)device_img[r * cols + c - 1]  + 2 * (float)device_img[r * cols + c + 1]
                       -(float)device_img[(r + 1) * cols + c - 1] + (float)device_img[(r + 1) * cols + c + 1];
        grad_x = abs(grad_x);

        float grad_y = -(float)device_img[(r - 1) * cols + c - 1] - 2 * (float)device_img[(r - 1) * cols + c] - (float)device_img[(r - 1) * cols + c + 1]
                       +(float)device_img[(r + 1) * cols + c - 1] + 2 * (float)device_img[(r + 1) * cols + c] + (float)device_img[(r + 1) * cols + c + 1];
        grad_y = abs(grad_y);

        float grad = grad_x / 2.0 + grad_y / 2.0;
        device_grad[r * cols + c] = grad;
    }

    else // set the boundary values to 0
    {
        device_grad[r * cols + c] = 0;
    }
}


__global__ void setup_rand_kernel(hiprandState *state)
{
    int idx = threadIdx.x * blockDim.x + threadIdx.y;
    hiprand_init(idx, 0, 0, &state[idx]);
}


__global__ void select_vertex_kernel(float* device_grad, Point* device_ownerMap, hiprandState *my_curandstate, float edgeThresh, float edgeP, float nonEdgeP, float boundP, int rows, int cols)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= cols || r >= rows)
        return;

    int idx = r * cols + c;
    int randIdx = threadIdx.x * blockDim.x + threadIdx.y;
    float randNum = hiprand_uniform(my_curandstate+randIdx);

    if (r > 0 && c > 0 && r < rows - 1 && c < cols - 1) // inside the image
    {
        if (device_grad[idx] > edgeThresh)
        {
            if (randNum <= edgeP)
            {
                Point p;
                p.x = c;
                p.y = r;
                device_ownerMap[idx] = p;
            }
        } else
        {
            if (randNum <= nonEdgeP)
            {
                Point p;
                p.x = c;
                p.y = r;
                device_ownerMap[idx] = p;
            }
        }
    }

    else // boundary
    {
        if (randNum <= boundP)
        {
            Point p;
            p.x = c;
            p.y = r;
            device_ownerMap[idx] = p;
        }
    }
}


void PrintDevice()
{
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}


void getGradGPU(cv::Mat &img)
{
    PrintDevice();

    double tolt_start = CycleTimer::currentSeconds();

    int rows = img.rows;
    int cols = img.cols;
    int numPixel = rows * cols;

    cv::Mat imgGray;
    imgGray.create(rows, cols, CV_8UC1);
    cv::cvtColor(img, imgGray, CV_BGR2GRAY);

    hipMalloc(&device_img, sizeof(uint8_t)*numPixel);    
    hipMalloc(&device_grad, sizeof(float)*numPixel);
    hipMemcpy(device_img, imgGray.data, sizeof(uint8_t)*numPixel, hipMemcpyHostToDevice);

    unsigned int n = 32;
    dim3 blockDim(n, n);
    dim3 gridDim((cols + n - 1) / n, (rows + n - 1) / n);

    double comp_start = CycleTimer::currentSeconds();

    get_grad_kernel<<<gridDim, blockDim>>>(device_img, device_grad, rows, cols);
    gpuErrchk(hipDeviceSynchronize());

    cout<<"Get grad GPU computation time: "<< (CycleTimer::currentSeconds() - comp_start) * 1000 <<"ms"<<endl;
    cout<<"Get grad GPU total time (include first malloc): "<< (CycleTimer::currentSeconds() - tolt_start) * 1000 <<"ms"<<endl;
}


void selectVerticesGPU(float edgeThresh, float edgeP, float nonEdgeP, float boundP, int rows, int cols)
{
    double comp_start = CycleTimer::currentSeconds();

    int numPixel = rows * cols;

    unsigned int n = 32;
    dim3 blockDim(n, n);
    dim3 gridDim((cols + n - 1) / n, (rows + n - 1) / n);

    hiprandState *device_state;
    hipMalloc(&device_state, sizeof(hiprandState)*n*n);

    hipMalloc(&device_ownerMap, sizeof(Point)*numPixel);
    hipMemset(device_ownerMap, -1, sizeof(Point)*numPixel);

    setup_rand_kernel<<<1, blockDim>>>(device_state); // this hiprand_init is slow
    gpuErrchk(hipDeviceSynchronize());  
    select_vertex_kernel<<<gridDim, blockDim>>>(device_grad, device_ownerMap, device_state, edgeThresh, edgeP, nonEdgeP, boundP, rows, cols);
    gpuErrchk(hipDeviceSynchronize());   

    hipFree(device_grad);
    hipFree(device_state);

    cout<<"Select vertices GPU time: "<< (CycleTimer::currentSeconds() - comp_start) * 1000 <<"ms"<<endl;
}


vector<Triangle> DelauneyGPU(int rows, int cols)
{
    // define grid and block size
    unsigned int n = 32;
    dim3 blockDim(n, n);
    dim3 gridDim((cols + n - 1) / n, (rows + n - 1) / n);

    double comp_start = CycleTimer::currentSeconds();

    // Step1 : find Voronoi graph
    int start_stepsize = NextPower2_CPU(min(rows, cols)) / 2;
    for(int stepsize = start_stepsize; stepsize>=1; stepsize /= 2)
    {
        voronoi_kernel<<<gridDim, blockDim>>>(device_ownerMap, stepsize, rows, cols);
        gpuErrchk(hipDeviceSynchronize());
    }

    // Step 2: find the number of triangles
    int* device_triangle_cnts;
    hipMalloc(&device_triangle_cnts, sizeof(int) * rows * cols);

    count_triangle_kernel<<<gridDim, blockDim>>>(device_ownerMap, rows, cols, device_triangle_cnts);
    gpuErrchk(hipDeviceSynchronize());

    // Step 3: prefix sum #triangles
    int* device_sum_triangles;
    hipMalloc(&device_sum_triangles, sizeof(int) * rows * cols);
    thrust::inclusive_scan(thrust::device, device_triangle_cnts, device_triangle_cnts + rows*cols, device_sum_triangles);

    // Step 4: build the triangles
    Triangle* device_triangles;
    int num_triangles;
    hipMemcpy(&num_triangles, &device_sum_triangles[rows*cols-1], sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&device_triangles, sizeof(Triangle) * num_triangles);
    triangle_kernel<<<gridDim, blockDim>>>(device_ownerMap, device_triangles, rows, cols, device_sum_triangles);
    gpuErrchk(hipDeviceSynchronize());

    // copy triangle data to CPU
    Triangle triangles[num_triangles];
    hipMemcpy(triangles, device_triangles, sizeof(Triangle)*num_triangles, hipMemcpyDeviceToHost);

    vector<Triangle> ret(triangles, triangles + num_triangles);

    // free
    hipFree(device_triangle_cnts);
    hipFree(device_sum_triangles);
    hipFree(device_triangles);

    cout<<"Delauney Core computation time: "<< (CycleTimer::currentSeconds() - comp_start) * 1000 <<"ms"<<endl;

    return ret;
}
